#include "hip/hip_runtime.h"
extern "C" {

__device__ float lap(float shar[14][14][14], float ddm2, int x, int y, int z, int ox, int oy, int oz, int c){
    int m = c - 1;
    int p = c + 1;
    return ddm2 *
        (shar[x + ox + m][y + oy + c][z + oz + c] + shar[x + ox + p][y + oy + c][z + oz + c]
         + shar[x + ox + c][y + oy + m][z + oz + c] + shar[x + ox + c][y + oy + p][z + oz + c]
         + shar[x + ox + c][y + oy + c][z + oz + m] + shar[x + ox + c][y + oy + c][z + oz + p]
         - 6.0f * shar[x + ox + c][y + oy + c][z + oz + c]);
}

__global__ void kernel_timestep(float* in, float* out, int sx, int sy, int sz, float ddm2){
    //    ddm2 = 1.0f;

    __shared__ float cache     [14][14][14];
    __shared__ float cache_lap [14][14][14];  // 12x12x12
    __shared__ float cache_lap2[14][14][14]; // 10x10x10
    __shared__ float cache_lap3[14][14][14]; // 8x8x8

    int sq14 = 14 * 14;

    int x = threadIdx.x;
    int y = threadIdx.y;
    int z = threadIdx.z;

    int bx = blockIdx.x * 8;
    int by = blockIdx.y * 8;
    int bz = blockIdx.z * 8;

    int load_idx = (z * 8 + y) * 8 + x;
    int load_y = load_idx / 14;
    int load_z = load_idx % 14;

    int base_addr = ((bz + load_z) * sy + by + load_y) * sx + bx;

    if(load_idx < sq14){
        for(int load_x = 0; load_x < 14; load_x++){
            cache[load_x][load_y][load_z] = in[base_addr + load_x];
        }
    }

    __syncthreads();
    // cache is filled now

    // ********************************************************************************
    // compute lap  
    // phase 0,0,0
    int ox = 0; int oy = 0; int oz = 0;
    cache_lap[x + ox + 1][y + oy + 1][z + oz + 1]                             = lap(cache, ddm2, x, y, z, ox, oy, oz, 1);
    __syncthreads();
    // phase x,0,0
    ox = 8; oy = 0; oz = 0;
    if(x < 4) cache_lap[x + ox + 1][y + oy + 1][z + oz + 1]                   = lap(cache, ddm2, x, y, z, ox, oy, oz, 1);
    __syncthreads();
    // phase 0,y,0
    ox = 0; oy = 8; oz = 0;
    if(y < 4) cache_lap[x + ox + 1][y + oy + 1][z + oz + 1]                   = lap(cache, ddm2, x, y, z, ox, oy, oz, 1);
    __syncthreads();
    // phase 0,0,z
    ox = 0; oy = 0; oz = 8;
    if(z < 4) cache_lap[x + ox + 1][y + oy + 1][z + oz + 1]                   = lap(cache, ddm2, x, y, z, ox, oy, oz, 1);
    __syncthreads();
    // phase x,y,0
    ox = 8; oy = 8; oz = 0;
    if(x < 4 && y < 4) cache_lap[x + ox + 1][y + oy + 1][z + oz + 1]          = lap(cache, ddm2, x, y, z, ox, oy, oz, 1);
    __syncthreads();
    // phase x,0,z
    ox = 8; oy = 0; oz = 8;
    if(x < 4 && z < 4) cache_lap[x + ox + 1][y + oy + 1][z + oz + 1]          = lap(cache, ddm2, x, y, z, ox, oy, oz, 1);
    __syncthreads();
    // phase 0,y,z
    ox = 0; oy = 8; oz = 8;
    if(y < 4 && z < 4) cache_lap[x + ox + 1][y + oy + 1][z + oz + 1]          = lap(cache, ddm2, x, y, z, ox, oy, oz, 1);
    __syncthreads();
    // phase x,y,z
    ox = 8; oy = 8; oz = 8;
    if(x < 4 && y < 4 && z < 4) cache_lap[x + ox + 1][y + oy + 1][z + oz + 1] = lap(cache, ddm2, x, y, z, ox, oy, oz, 1);
    __syncthreads();
    // ********************************************************************************


    // ********************************************************************************
    // compute lap2
    // phase 0,0,0
    ox = 0; oy = 0; oz = 0;
    cache_lap2[x + ox + 2][y + oy + 2][z + oz + 2]                             = lap(cache_lap, ddm2, x, y, z, ox, oy, oz, 2);
    __syncthreads();
    // phase x,0,0
    ox = 8; oy = 0; oz = 0;
    if(x < 2) cache_lap2[x + ox + 2][y + oy + 2][z + oz + 2]                   = lap(cache_lap, ddm2, x, y, z, ox, oy, oz, 2);
    __syncthreads();
    // phase 0,y,0
    ox = 0; oy = 8; oz = 0;
    if(y < 2) cache_lap2[x + ox + 2][y + oy + 2][z + oz + 2]                   = lap(cache_lap, ddm2, x, y, z, ox, oy, oz, 2);
    __syncthreads();
    // phase 0,0,z
    ox = 0; oy = 0; oz = 8;
    if(z < 2) cache_lap2[x + ox + 2][y + oy + 2][z + oz + 2]                   = lap(cache_lap, ddm2, x, y, z, ox, oy, oz, 2);
    __syncthreads();
    // phase x,y,0
    ox = 8; oy = 8; oz = 0;
    if(x < 2 && y < 2) cache_lap2[x + ox + 2][y + oy + 2][z + oz + 2]          = lap(cache_lap, ddm2, x, y, z, ox, oy, oz, 2);
    __syncthreads();
    // phase x,0,z
    ox = 8; oy = 0; oz = 8;
    if(x < 2 && z < 2) cache_lap2[x + ox + 2][y + oy + 2][z + oz + 2]          = lap(cache_lap, ddm2, x, y, z, ox, oy, oz, 2);
    __syncthreads();
    // phase 0,y,z
    ox = 0; oy = 8; oz = 8;
    if(y < 2 && z < 2) cache_lap2[x + ox + 2][y + oy + 2][z + oz + 2]          = lap(cache_lap, ddm2, x, y, z, ox, oy, oz, 2);
    __syncthreads();
    // phase x,y,z
    ox = 8; oy = 8; oz = 8;
    if(x < 2 && y < 2 && z < 2) cache_lap2[x + ox + 2][y + oy + 2][z + oz + 2] = lap(cache_lap, ddm2, x, y, z, ox, oy, oz, 2);
    __syncthreads();
    // ********************************************************************************

    // ********************************************************************************
    // compute lap3
    cache_lap3[x + 3][y + 3][z + 3] = lap(cache_lap2, ddm2, x, y, z, 0, 0, 0, 3);
    __syncthreads();
    // ********************************************************************************

    float c = cache[x + 3][y + 3][z + 3];
    float xm = cache[x + 2][y + 3][z + 3];
    float xp = cache[x + 4][y + 3][z + 3];
    float ym = cache[x + 3][y + 2][z + 3];
    float yp = cache[x + 3][y + 4][z + 3];
    float zm = cache[x + 3][y + 3][z + 2];
    float zp = cache[x + 3][y + 3][z + 4];

    float lapPsi3 = ddm2 *
        (xm * xm * xm + xp * xp * xp
         + ym * ym * ym + yp * yp * yp
         + zm * zm * zm + zp * zp * zp
         - 6.0f * c * c * c);

    // ********************************************************************************
    // PFC
    out[((bz + z + 3) * sy + by + y + 3) * sx + bx + x + 3] = 
        cache[x + 3][y + 3][z + 3]
        + 0.0001f * (lapPsi3
                + (1.0f - 0.25f) * cache_lap[x + 3][y + 3][z + 3]
                + 2.0f * cache_lap2[x + 3][y + 3][z + 3]
                + cache_lap3[x + 3][y + 3][z + 3]);
    // ********************************************************************************

    // ********************************************************************************
    // LAPLACE
    //    out[((bz + z + 3) * sy + by + y + 3) * sx + bx + x + 3] = 0.16666666666f * (cache_lap[x + 3][y + 3][z + 3] + 6.0f * cache[x + 3][y + 3][z + 3]);
    // ********************************************************************************

    // ********************************************************************************
    // ID
    // out[((bz + z + 3) * sy + by + y + 3) * sx + bx + x + 3] = cache_lap[x + 3][y + 3][z + 3];
    // ********************************************************************************

    // ********************************************************************************
    // ID
    // out[((bz + z + 3) * sy + by + y + 3) * sx + bx + x + 3] = in[((bz + z + 3) * sy + by + y + 3) * sx + bx + x + 3];
    // ********************************************************************************
}



__device__ int ifun(int sx, int sy, int sz, int x, int y, int z){ return (z * sy + y) * sx + x; }
#define I(x,y,z) ifun(sx, sy, sz, x, y, z)
__device__ int mod(float a, float b){ return ((int)a) - ((int)b) * floor(a / b); }
__device__ int wrap(int a, int b){ return mod(a - 3, b) + 3; }

__global__ void kernel_pbc_noz(float* ar, int sx, int sy, int sz){
    int sxp = sx - 6;
    int syp = sy - 6;
    int szp = sz - 6;
    int a = threadIdx.x;
    int b = threadIdx.y;
    // threadblock: 16x16
    int x_mul_max = sx / 16;
    int y_mul_max = sy / 16;
    int z_mul_max = sz / 16;

    // x - y
    // for(int x_mul = 0; x_mul <= x_mul_max; x_mul++){
    //   for(int y_mul = 0; y_mul <= y_mul_max; y_mul++){
    //     int xx = 16 * x_mul + a;
    //     int yy = 16 * y_mul + b;
    //     if(xx < sx && yy < sy){
    // 	int xc = wrap(xx, sxp);
    // 	int yc = wrap(yy, syp);
    // 	ar[I(xx, yy, sz - 3)] = ar[I(xc, yc, 3)];
    // 	ar[I(xx, yy, sz - 2)] = ar[I(xc, yc, 4)];
    // 	ar[I(xx, yy, sz - 1)] = ar[I(xc, yc, 5)];
    // 	ar[I(xx, yy, 2)] = ar[I(xc, yc, sz - 4)];
    // 	ar[I(xx, yy, 1)] = ar[I(xc, yc, sz - 5)];
    // 	ar[I(xx, yy, 0)] = ar[I(xc, yc, sz - 6)];
    //     }
    //   }
    // }  

    // x - z
    for(int x_mul = 0; x_mul <= x_mul_max; x_mul++){
        for(int z_mul = 0; z_mul <= z_mul_max; z_mul++){
            int xx = 16 * x_mul + a;
            int zz = 16 * z_mul + b;
            if(xx < sx && zz < sz){
                int xc = wrap(xx, sxp);
                int zc = wrap(zz, szp);
                ar[I(xx, sy - 3, zz)] = ar[I(xc, 3, zc)];
                ar[I(xx, sy - 2, zz)] = ar[I(xc, 4, zc)];
                ar[I(xx, sy - 1, zz)] = ar[I(xc, 5, zc)];
                ar[I(xx, 2, zz)] = ar[I(xc, sy - 4, zc)];
                ar[I(xx, 1, zz)] = ar[I(xc, sy - 5, zc)];
                ar[I(xx, 0, zz)] = ar[I(xc, sy - 6, zc)];
            }
        }
    }

    // y - z
    for(int y_mul = 0; y_mul <= y_mul_max; y_mul++){
        for(int z_mul = 0; z_mul <= z_mul_max; z_mul++){
            int yy = 16 * y_mul + a;
            int zz = 16 * z_mul + b;
            if(yy < sy && zz < sz){
                int yc = wrap(yy, syp);
                int zc = wrap(zz, szp);
                ar[I(sx - 3, yy, zz)] = ar[I(3, yc, zc)];
                ar[I(sx - 2, yy, zz)] = ar[I(4, yc, zc)];
                ar[I(sx - 1, yy, zz)] = ar[I(5, yc, zc)];
                ar[I(2, yy, zz)] = ar[I(sx - 4, yc, zc)];
                ar[I(1, yy, zz)] = ar[I(sx - 5, yc, zc)];
                ar[I(0, yy, zz)] = ar[I(sx - 6, yc, zc)];
            }
        }
    }
}

__global__ void kernel_ghost_copy(float* ar, float* out_left, float* out_right, int sx, int sy, int sz){
    int y_mul_max = sy / 256;
    for(int z = 0; z < 3; z++){
        for(int y_mul = 0; y_mul <= y_mul_max; y_mul++){
            int y = y_mul * 256 + threadIdx.x;
            if(y < sy){
                int base_addr_lin = (z * sy + y) * sx;
                int base_addr_left = ((z + 3) * sy + y) * sx;
                int base_addr_right = ((sz - 6 + z) * sy + y) * sx;
                for(int x = 0; x < sx; x++){
                    out_left[base_addr_lin + x] = ar[base_addr_left + x];
                    out_right[base_addr_lin + x] = ar[base_addr_right + x];
                }
            }
        }
    }
}

__global__ void kernel_ghost_copy_inv(float* ar, float* out_left, float* out_right, int sx, int sy, int sz){
    int y_mul_max = sy / 256;
    for(int z = 0; z < 3; z++){
        for(int y_mul = 0; y_mul <= y_mul_max; y_mul++){
            int y = y_mul * 256 + threadIdx.x;
            if(y < sy){
                int base_addr_lin = (z * sy + y) * sx;
                int base_addr_left = ((z + 0) * sy + y) * sx;
                int base_addr_right = ((sz - 3 + z) * sy + y) * sx;
                for(int x = 0; x < sx; x++){
                    ar[base_addr_left + x] = out_left[base_addr_lin + x];
                    ar[base_addr_right + x] = out_right[base_addr_lin + x];
                }
            }
        }
    }
}

  __global__ void kernel_source(float* ar, int sx, int sy, int sz){
    ar[((3) * sy + 3) * sx + 3] = 1.0f;
    //    ar[((3 + 200) * sy + 3) * sx + 3 + 1200] = 1.0f;
  }
}

} // extern "C"
